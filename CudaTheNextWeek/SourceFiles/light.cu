#include "hip/hip_runtime.h"
#include "header.cuh"

#define SPHERE_COUNT 2
#define RECTAGLE_COUNT 1

#define OBJECT_COUNT SPHERE_COUNT + RECTAGLE_COUNT

/* Iterative ray color function
 * Recursive call woulde be:
    return emitted + attenuarion * rec_call
 * So, general recursive formula would be:
 * r(1) = e1 + a1 * r(0)
 * r(2) = e2 + a2 * r(1) => r(2) = e2 + a2 * (e1 + a1 * r(0)) => r(2) = e2 + a2 * e1 + a2 * a1 * r(0)
 * ...
 * r(i) = an * ... * a2 * a1 * r(0) + SIGMA(i = 1, n) e(i) * a(i + 1) * ... * a(n)
* So we use an iterative approach
* Step i would be:
    curAttenuation = curAttenuation * attenuation
    curEmitted = curEmitted + curAttenuation * emitted

*/
__device__ color rayColor(const ray& r, const color& background, hittable **world, hiprandState *localRandState) {
    ray curRay = r;

    color curAttenuation(1.0f, 1.0f, 1.0f);
    color curEmitted(0.0f, 0.0f, 0.0f);

    for (int i = 0; i < 50; i ++) {
        hit_record rec;
        if ((*world)->hit(curRay, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            color attenuation;
            color emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            if (rec.mat_ptr->scatter(curRay, rec, attenuation, scattered, localRandState)) {
                curEmitted = curEmitted + curAttenuation * emitted;
                curAttenuation = curAttenuation * attenuation;
                curRay = scattered;
            } else {
                return curAttenuation * emitted + curEmitted;
            }
       } else {
              return curAttenuation * background + curEmitted;
        }
   }
   // exceeded recursion
   return color(0.0f, 0.0f, 0.0f);
}

__global__ void renderInit(int maxX, int maxY, hiprandState *randStatePixels, hiprandState *randStateWorld) {
    // Also initialize here the random state for world construction
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, randStateWorld);
    }

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    int pixelIndex = j * maxX + i;

    // Aperently we get better results if we use a different seed for each pixel
    // and same sequence for each thread
    hiprand_init(1984 + pixelIndex, 0, 0, &randStatePixels[pixelIndex]);
}

__global__ void render(vec3 *fb, int maxX, int maxY, int ns, camera **cam, hittable **world, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    int pixelIndex = j * maxX + i;
    hiprandState localRandState = randState[pixelIndex];
    color pixelColor(0.0f, 0.0f, 0.0f);
    color background(0.0f, 0.0f, 0.0f);
    for (int s = 0; s < ns; s ++) {
        float u = float(i + hiprand_uniform(&localRandState)) / float(maxX);
        float v = float(j + hiprand_uniform(&localRandState)) / float(maxY);

        ray r = (*cam)->get_ray(u, v, &localRandState);
        pixelColor += rayColor(r, background, world, &localRandState);
    }

    getColor(pixelColor, ns);

    fb[pixelIndex] = pixelColor;
}

__global__ void allocateWorld(hittable **d_list, hittable **d_world, camera **d_cam, hiprandState *randState) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_world) = new hittable_list(d_list, OBJECT_COUNT);

        noise_texture *perlinTexture = new noise_texture(randState, 4.0f);

        *(d_list) = new moving_sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f, new lambertian(perlinTexture));
        *(d_list + 1) = new moving_sphere(vec3(0.0f, 2.0f, 0.0f), 2.0f, new lambertian(perlinTexture));

        diffuse_light *light = new diffuse_light(new constant_texture(vec3(4.0f, 4.0f, 4.0f)));
        *(d_list + 2) = new xy_rect(3.0f, 5.0f, 1.0f, 3.0f, -2.0f, light);

        // Camera
        vec3 lookFrom(26.0f, 3.0f, 6.0f);
        vec3 lookAt(0.0f, 2.0f, 0.0f);
        vec3 vUp(0.0f, 1.0f, 0.0f);
        float vfov = 20.0f;

        float distToFocus = 10.0f;
        float aperture = 0.1f;
        float aspect_ratio = 3.0f / 2.0f;

        *d_cam = new camera(lookFrom, lookAt, vUp, vfov, aspect_ratio, aperture, distToFocus, 0.0, 1.0);
    }
}

__global__ void freeWorld(hittable **d_list, hittable **d_world, camera **d_cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < SPHERE_COUNT; i ++) {
            delete ((moving_sphere *)d_list[i])->mat_ptr;
            delete *(d_list + i);
        }
        for (int i = SPHERE_COUNT; i < OBJECT_COUNT; i ++) {
            delete ((xy_rect *)d_list[i])->mp;
            delete *(d_list + i);
        }
        delete *(d_world);
        delete *(d_cam);
    }
}

int main(void) {
    int nx = 1200;
    int ny = 800;
    int ns = 400;

    int num_pixels = nx * ny;

    color *fb_gpu;
    hipError_t cudaStatus;

    // create device frame buffer
    cudaStatus = hipMalloc((void**)&fb_gpu, num_pixels * sizeof(color));
    checkReturn(cudaStatus);

    // create random state for each pixel
    hiprandState *d_randState;
    cudaStatus = hipMalloc((void**)&d_randState, num_pixels * sizeof(hiprandState));
    checkReturn(cudaStatus);

    // create random state for world construction
    hiprandState *d_worldRandState;
    cudaStatus = hipMalloc((void**)&d_worldRandState, sizeof(hiprandState));
    checkReturn(cudaStatus);

    // create world of hittable objects
    hittable **d_list;
    cudaStatus = hipMalloc((void**)&d_list, OBJECT_COUNT * sizeof(hittable*));
    checkReturn(cudaStatus);

    hittable **d_world;
    cudaStatus = hipMalloc((void**)&d_world, sizeof(hittable*));
    checkReturn(cudaStatus);

    // create camera
    camera **d_cam;
    cudaStatus = hipMalloc((void**)&d_cam, sizeof(camera*));
    checkReturn(cudaStatus);

    dim3 blockCount(nx + TX - 1 / TX, ny + TY - 1 / TY);
    dim3 blockSize(TX, TY);

    renderInit<<<blockCount, blockSize>>>(nx, ny, d_randState, d_worldRandState);
    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    allocateWorld<<<1, 1>>>(d_list, d_world, d_cam, d_worldRandState);
    checkReturn(hipGetLastError());

    // Create events until world is created
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    checkReturn(hipDeviceSynchronize());
    checkReturn(hipEventRecord(start));

    render<<<blockCount, blockSize>>>(fb_gpu, nx, ny, ns, d_cam, d_world, d_randState);

    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    checkReturn(hipEventRecord(stop));

    float milliseconds = 0;
    checkReturn(hipEventElapsedTime(&milliseconds, start, stop));
    std::cerr << "Elapsed time: " << milliseconds << " ms\n";

    color *fb_cpu = (color*)malloc(num_pixels * sizeof(color));
    cudaStatus = hipMemcpy(fb_cpu, fb_gpu, num_pixels * sizeof(color), hipMemcpyDeviceToHost);
    checkReturn(cudaStatus);

    // Output FB as Image
    std::ofstream ppmFile("light.ppm");

    ppmFile << "P3\n" << nx << " " << ny << "\n255\n";

    for (int j = ny - 1; j >= 0; j--) {
        std::cerr << "\rScanlines remaining: " << j << " " << std::flush;
        for (int i = 0; i < nx; i++) {
            size_t pixelIndex = j * nx + i;
            int ir = static_cast<int>(fb_cpu[pixelIndex].e[0]);
            int ig = static_cast<int>(fb_cpu[pixelIndex].e[1]);
            int ib = static_cast<int>(fb_cpu[pixelIndex].e[2]);
            ppmFile << ir << " " << ig << " " << ib << "\n";
    }
}

    // free world of hittable objects
    freeWorld<<<1, 1>>>(d_list, d_world, d_cam);
    checkReturn(hipGetLastError());

    cudaStatus = hipFree(d_list);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(d_world);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(fb_gpu);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(d_cam);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(d_randState);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(d_worldRandState);
    checkReturn(cudaStatus);

    free(fb_cpu);
    std::cerr << "\nDone.\n";
}