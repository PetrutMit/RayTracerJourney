#include "hip/hip_runtime.h"
#include "HeaderFiles/header.cuh"

#include "HeaderFiles/random.cuh"
#include "HeaderFiles/bvh.cuh"

#include "HeaderFiles/render.cuh"

#include <glm/packing.hpp>

#define NODE_COUNT 6
#define FILTER_SIZE 8

__device__ color rayColor(const ray& r, const color& background, hittable_list **world, hiprandState *localRandState, GBufferTexel *gBuffer) {
    ray curRay = r;

    color curAttenuation(1.0f, 1.0f, 1.0f);
    color curEmitted(0.0f, 0.0f, 0.0f);
    hit_record rec;

    for (int i = 0; i < 20; i ++) {
        if ((*world)->hit(curRay, interval(0.001, INF), rec)) {
            ray scattered;
            color attenuation;
            color emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            if (rec.mat_ptr->scatter(curRay, rec, attenuation, scattered, localRandState)) {
                curEmitted = curEmitted + curAttenuation * emitted;
                curAttenuation = curAttenuation * attenuation;
                curRay = scattered;

                // Populate gBuffer on depth 0
                if (i == 0) {
                    gBuffer->normal = rec.normal;
                    gBuffer->position = rec.p;
                }
            } else {
                return curAttenuation * emitted + curEmitted;
            }
       } else {
              return curAttenuation * background + curEmitted;
        }
   }
   // exceeded recursion
   return color(0.0f, 0.0f, 0.0f);
}

__global__ void renderInit(int maxX, int maxY, hiprandState *randStatePixels, hiprandState *randStateWorld) {
    // Also initialize here the random state for world construction
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, randStateWorld);
    }

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    int pixelIndex = j * maxX + i;

    // Aperently we get better results if we use a different seed for each pixel
    // and same sequence for each thread
    hiprand_init(1984 + pixelIndex, 0, 0, &randStatePixels[pixelIndex]);
}

__global__ void raytrace(int frame, vec3 *fbColor, int maxX, int maxY, int ns, camera **cam,
                         hittable_list **world, hiprandState *randState, float deltaTime, GBufferTexel *gBuffer) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    // Adjust our camera view
    if (i == 0 && j == 0) {
        (*cam)->adjust_parameters(deltaTime);
    }

    int pixelIndex = j * maxX + i;
    hiprandState localRandState = randState[pixelIndex];
    color pixelColor(0.0f, 0.0f, 0.0f);
    color background(0.2f, 0.2f, 0.2f);

    // Just one sample per pixel
    float u = float(i + hiprand_uniform(&localRandState)) / float(maxX);
    float v = float(j + hiprand_uniform(&localRandState)) / float(maxY);

    ray r = (*cam)->get_ray(u, v, &localRandState);
    pixelColor += rayColor(r, background, world, &localRandState, &gBuffer[pixelIndex]);

    getColor(pixelColor, ns);

    fbColor[pixelIndex] = pixelColor;
}

__global__ void atrousDenoise(GBufferTexel* gBuffer, int stepWidth, vec3 *rayTracedInput, vec3 *denoisedOutput, uint32_t *fb) {
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;

   if ((i >= 800) || (j >= 600)) return;

   const float c_phi = 1.45f;
   const float n_phi = 1.30f;
   const float p_phi = 1.25f;

   static constexpr float kernel[] = { 3.f / 8.f, 1.f / 4.f, 1.f / 16.f };
    
   int pixelIndex = j * 800 + i;
   GBufferTexel center = gBuffer[pixelIndex];
   vec3 center_normal = center.normal;
   vec3 center_position = center.position;
   vec3 center_albedo = rayTracedInput[pixelIndex];

   vec3 sum_albedo(0.0f);
   float sum_weight = 0.0f;
   
   for (int dy = -2; dy <= 2; dy++) {
       for (int dx = -2; dx <= 2; dx++) {
           const int u = glm::clamp(i + dx * stepWidth, 0, 800);
           const int v = glm::clamp(j + dy * stepWidth, 0, 600);

           const int index = v * 800 + u;
		   const GBufferTexel& texel = gBuffer[index];

           const vec3 normal = texel.normal;
           const vec3 position = texel.position;
           const vec3 albedo = rayTracedInput[index];

		   vec3 diff = center_position - position;
           float dist = diff.length_squared();
           const float p_weight = fminf(std::exp(-dist / p_phi), 1.0f);

           diff = center_normal - normal;
           dist = diff.length_squared();
           const float n_weight = fminf(std::exp(-dist / n_phi), 1.0f);

           diff = center_albedo - albedo;
           dist = diff.length_squared();
           const float c_weight = fminf(std::exp(-dist / c_phi), 1.0f);

           const float weight = p_weight * n_weight * c_weight;

           const int kernel_index = fminf(std::abs(dx), std::abs(dy));
		   sum_albedo += albedo * kernel[kernel_index] * weight;

           sum_weight += kernel[kernel_index] * weight;
       }
    }

   vec3 denoisedPixel = sum_albedo / sum_weight;
   denoisedOutput[pixelIndex] = denoisedPixel;
    // On the last iteration, write the pixel to OpenGL bound buffer
    if (stepWidth == FILTER_SIZE) {
        fb[pixelIndex] = glm::packUnorm4x8(glm::vec4(denoisedPixel.z(), denoisedPixel.y(), denoisedPixel.x(), 1.0f));
    }
}

__global__ void allocateWorld(hittable **d_list, hittable_list **d_world, camera **d_cam, hiprandState *randState) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {

        lambertian *ground = new lambertian(color(0.83f, 0.83f, 0.13f));
        *(d_list) = new sphere(vec3(0.0f, -2000.0f, 0.0f), 2000.0f, ground);

        diffuse_light *light = new diffuse_light(color(3.0f, 3.0f, 3.0f));
        sphere *moon = new sphere(vec3(-550.0f, 350.0f, 550.0f), 50.0f, light);
        *(d_list + 1) = moon; 

        *(d_list + 2) = new sphere(vec3(-80.0f, 20.0f, -300.0f), 40.0f, new metal(color(0.1f, 0.9f, 0.1f), 0.1f));

        hittable **spheres = new hittable*[64];
        lambertian *red = new lambertian(color(0.9f, 0.2f, 0.1f));
        for (int i = 0; i < 64; i ++) {
            spheres[i] = new sphere(vec3(randomVectorBetween(randState, -50.0f, 0.0f) + vec3(50.0f, 30.0f, -300.0f)), 7.0f, red);
        }

        *(d_list + 3) = new bvh_node(spheres, 64, randState);

        // Create some random height boxes
        hittable **boxes = new hittable*[4];

        float width = 20.0f;
        int choice;
        float x0, z0, x1, z1, y0, y1;
        material *mat;
        for (int i = 0; i < 4; i ++) {
            x0 = 80.0f + i * width;
            z0 = -300.0f;

            x1 = x0 + width;
            z1 = -300.0f + width;
    
            y0 = -30.0f;
            y1 = randomFloat(randState, 1.0f, 100.0f);
            
            choice = randomInt(randState, 0, 2);
            printf("choice: %d\n", choice);
            switch (choice)
            {
            case 0:
                mat = new lambertian(randomVector(randState));
                break;
            case 1:
                mat = new metal(randomVector(randState), randomFloat(randState, 0.0f, 0.3f));
                break;
            }
            boxes[i] = box(vec3(x0, y0, z0), vec3(x1, y1, z1), mat);
		}

        *(d_list + 4) = new bvh_node(boxes, 4, randState);

        // Create random spheres on the ground
        hittable **ground_spheres = new hittable*[64];
        int cnt = 0;
        float chooseMat;
        float randomRadius;

        for (float i = -400.0f; i < 400.0f; i += 100.0f) {
			for (int j = -400.0f; j < 400.0f; j += 100.0f) {
				chooseMat = hiprand_uniform(randState);
                randomRadius = randomFloat(randState, 3.0f, 10.0f);

				vec3 center(i + 100.0f * hiprand_uniform(randState), 0.0f, j + 100.0f * hiprand_uniform(randState));
			    if (chooseMat < 0.8f) {
                    ground_spheres[cnt++] = new sphere(center, randomRadius, new lambertian(randomVector(randState)));
			    } else if (chooseMat < 0.95f) {
				    ground_spheres[cnt++] = new sphere(center, randomRadius, new metal(randomVector(randState), randomFloat(randState, 0.0f, 0.5f)));
			    } else {
				    ground_spheres[cnt++] = new sphere(center, randomRadius, new dielectric(1.5f));
			    }
			}
		}

        *(d_list + 5) = new bvh_node(ground_spheres, 64, randState);

        *(d_world) = new hittable_list(d_list, NODE_COUNT);

        vec3 lookFrom(0.0f, 0.0f, -600.0f);
        vec3 lookAt(0.0f, 0.0f, 0.0f);
        float distToFocus = 10.0f;
        float aperture = 0.0f;
        float aspect_ratio = 3.0f / 2.0f;
        float vfov = 40.0f;

        *(d_cam) = new camera(lookFrom, lookAt, vec3(0.0f, 1.0f, 0.0f), vfov, aspect_ratio, aperture, distToFocus);
    }
}

__global__ void freeWorld(hittable **d_list, hittable_list **d_world, camera **d_cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < NODE_COUNT; i++) {
            delete *(d_list + i);
        }
        delete *(d_world);
        delete *(d_cam);
    }
}


Render::Render(int nx, int ny, hipGraphicsResource_t cuda_pbo_resource) {
    _nx = nx;
    _ny = ny;

    int num_pixels = _nx * _ny;

    hipError_t cudaStatus;

    // create random state for each pixel
    hiprandState *d_randStatePixels;
    cudaStatus = hipMalloc((void**)&d_randStatePixels, num_pixels * sizeof(hiprandState));
    checkReturn(cudaStatus);

    // create random state for world construction
    hiprandState *d_randStateWorld;
    cudaStatus = hipMalloc((void**)&d_randStateWorld, sizeof(hiprandState));
    checkReturn(cudaStatus);

    // create world of hittable objects
    hittable **d_list;
    cudaStatus = hipMalloc((void**)&d_list, NODE_COUNT * sizeof(hittable*));
    checkReturn(cudaStatus);

    hittable_list **d_world;
    cudaStatus = hipMalloc((void**)&d_world, sizeof(hittable_list*));
    checkReturn(cudaStatus);

    // create camera
    camera **d_cam;
    cudaStatus = hipMalloc((void**)&d_cam, sizeof(camera*));
    checkReturn(cudaStatus);

    // create gBuffer
    GBufferTexel *d_gbuffer;
    cudaStatus = hipMalloc((void**)&d_gbuffer, num_pixels * sizeof(GBufferTexel));
    checkReturn(cudaStatus);

    // create output color for raytracing
    cudaStatus = hipMalloc((void**)&_d_rayTracedImage, num_pixels * sizeof(vec3));
    checkReturn(cudaStatus);

    // create output color for denoising
    cudaStatus = hipMalloc((void**)&_d_denoisedImage, num_pixels * sizeof(vec3));
    checkReturn(cudaStatus);

    // Even though we have an iterative approach, we still need a bigger stack
    size_t size;
    hipDeviceGetLimit(&size, hipLimitStackSize);
    hipDeviceSetLimit(hipLimitStackSize, 2 * size);

    dim3 blockCount(_nx + TX - 1 / TX, _ny + TY - 1 / TY);
    dim3 blockSize(TX, TY);

    renderInit<<<blockCount, blockSize>>>(_nx, _ny, d_randStatePixels, d_randStateWorld);
    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    allocateWorld<<<1, 1>>>(d_list, d_world, d_cam, d_randStateWorld);
    checkReturn(hipGetLastError());

    // Free random state for world construction
    hipFree(d_randStateWorld);

    _d_cam = d_cam;
    _d_world = d_world;
    _d_randStatePixels = d_randStatePixels;
    _d_gBuffer = d_gbuffer;

    _cuda_pbo_resource = cuda_pbo_resource;
    cudaStatus = hipGraphicsMapResources(1, &_cuda_pbo_resource);
    checkReturn(cudaStatus);

    cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&_d_output, NULL, _cuda_pbo_resource);
    checkReturn(cudaStatus);
}

__host__ void Render::render(float deltaTime, int frame) {
    dim3 blockCount(_nx + TX - 1 / TX, _ny + TY - 1 / TY);
    dim3 blockSize(TX, TY);
    int ns = 1;

    raytrace<<<blockCount, blockSize>>>(frame, _d_rayTracedImage, _nx, _ny, ns, _d_cam, _d_world, _d_randStatePixels, deltaTime, _d_gBuffer);
    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());
}

__host__ void Render::denoise() {
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (800 + blockSize2d.x - 1) / blockSize2d.x,
        (600 + blockSize2d.y - 1) / blockSize2d.y);
    const int pixelcount = 800 * 600;

    for (int stepWidth = 1; stepWidth <= FILTER_SIZE; stepWidth *= 2) {
        atrousDenoise<<<blocksPerGrid2d, blockSize2d>>>(_d_gBuffer, stepWidth, _d_rayTracedImage, _d_denoisedImage, _d_output);
        checkReturn(hipGetLastError());
        checkReturn(hipDeviceSynchronize());
        // Swap the buffers
        vec3 *temp = _d_rayTracedImage;
        _d_rayTracedImage = _d_denoisedImage;
        _d_denoisedImage = temp;
    }
}

Render::~Render() {
    freeWorld<<<1, 1>>>((*_d_world)->list, _d_world, _d_cam);
    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    hipFree(_d_randStatePixels);
    checkReturn(hipGraphicsUnmapResources(1, &_cuda_pbo_resource));
}
