#include "hip/hip_runtime.h"
#include "HeaderFiles/header.cuh"

#include "HeaderFiles/random.cuh"
#include "HeaderFiles/bvh.cuh"

#include "HeaderFiles/render.cuh"

#define NODE_COUNT 8

__device__ color Render::rayColor(const ray& r, const color& background, hittable **world, hiprandState *localRandState) {
    ray curRay = r;

    color curAttenuation(1.0f, 1.0f, 1.0f);
    color curEmitted(0.0f, 0.0f, 0.0f);
    hit_record rec;

    for (int i = 0; i < 20; i ++) {
        if ((*world)->hit(curRay, interval(0.001, INF), rec)) {
            ray scattered;
            color attenuation;
            color emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            if (rec.mat_ptr->scatter(curRay, rec, attenuation, scattered, localRandState)) {
                curEmitted = curEmitted + curAttenuation * emitted;
                curAttenuation = curAttenuation * attenuation;
                curRay = scattered;
            } else {
                return curAttenuation * emitted + curEmitted;
            }
       } else {
              return curAttenuation * background + curEmitted;
        }
   }
   // exceeded recursion
   return color(0.0f, 0.0f, 0.0f);
}

__global__ void Render::renderInit(int maxX, int maxY, hiprandState *randStatePixels, hiprandState *randStateWorld) {
    // Also initialize here the random state for world construction
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, randStateWorld);
    }

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    int pixelIndex = j * maxX + i;

    // Aperently we get better results if we use a different seed for each pixel
    // and same sequence for each thread
    hiprand_init(1984 + pixelIndex, 0, 0, &randStatePixels[pixelIndex]);
}

__global__ void Render::raytrace(vec3 *fb, int maxX, int maxY, int ns, camera **cam, hittable **world, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    int pixelIndex = j * maxX + i;
    hiprandState localRandState = randState[pixelIndex];
    color pixelColor(0.0f, 0.0f, 0.0f);
    color background(0.05f, 0.05f, 0.05f);
    for (int s = 0; s < ns; s ++) {
        float u = float(i + hiprand_uniform(&localRandState)) / float(maxX);
        float v = float(j + hiprand_uniform(&localRandState)) / float(maxY);

        ray r = (*cam)->get_ray(u, v, &localRandState);
        pixelColor += rayColor(r, background, world, &localRandState);
    }

    getColor(pixelColor, ns);

    fb[pixelIndex] = pixelColor;
}

__global__ void Render::allocateWorld(hittable **d_list, hittable **d_world, camera **d_cam, hiprandState *randState) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        lambertian *ground = new lambertian(color(0.48f, 0.83f, 0.53f));

        int boxes_per_side = 4;

        hittable **boxes = new hittable*[boxes_per_side * boxes_per_side];
        int cnt = 0;

        for (int i = 0; i < boxes_per_side; i ++) {
            for (int j = 0; j < boxes_per_side; j ++) {
                float w = 200.0f;
                float x0 = i * w;
                float z0 = j * w;
                float y0 = 0.0f;
                float x1 = x0 + w;
                float y1 = randomFloat(randState, 1.0f, 101.0f);
                float z1 = z0 + w;

                boxes[cnt ++] = box(vec3(x0, y0, z0), vec3(x1, y1, z1), ground);
            }
        }

        if (useBVH) {
            *(d_list) = new bvh_node(boxes, cnt, randState);
        } else {
            *(d_list) = new hittable_list(boxes, cnt);
        }

        diffuse_light *light = new diffuse_light(color(7.0f, 7.0f, 7.0f));
        quad *light_shape = new quad(vec3(123.0f, 554.0f, 147.0f), vec3(300.0f, 0.0f, 0.0f), vec3(0.0f, 0.0f, 265.0f), light);

        *(d_list + 1) = light_shape;

        vec3 center1 = vec3(400.0f, 400.0f, 200.0f);
        vec3 center2 = center1 + vec3(30.0f, 0.0f, 0.0f);

        *(d_list + 2) = new sphere(center1, center2, 50.0f, new lambertian(color(0.7f, 0.3f, 0.1f)));
        *(d_list + 3) = new sphere(vec3(260.0f, 150.0f, 45.0f), 50.0f, new dielectric(1.5f));
        *(d_list + 4) = new sphere(vec3(400.0f, 200.0f, 400.0f), 100.0f, new metal(color(0.8f, 0.8f, 0.9f), 10.0f));

        *(d_list + 5) = new sphere(vec3(360.0f, 150.0f, 145.0f), 70.0f, new metal(color(0.3f, 0.8f, 0.2f), 0.2f));

        noise_texture *pertext = new noise_texture(randState, 0.1f);
        *(d_list + 6) = new sphere(vec3(220.0f, 280.0f, 300.0f), 80.0f, new lambertian(pertext));

        hittable **spheres = new hittable*[1024];
        lambertian *blue = new lambertian(color(0.2f, 0.2f, 0.7f));
        for (int i = 0; i < 1024; i ++) {
            spheres[i] = new sphere(vec3(randomVectorBetween(randState, 0.0f, 165.0f)), 10.0f, blue);
        }

        if (useBVH) {
             *(d_list + 7) = new translate(new rotate_y(new bvh_node(spheres, 1024, randState), 15.0f), vec3(-100.0f, 270.0f, 395.0f));
         } else {
             *(d_list + 7) = new translate(new rotate_y(new hittable_list(spheres, 1024), 15.0f), vec3(-100.0f, 270.0f, 395.0f));
        }

        *(d_world) = new hittable_list(d_list, NODE_COUNT);

        vec3 lookFrom(478.0f, 278.0f, -600.0f);
        vec3 lookAt(278.0f, 278.0f, 0.0f);
        float distToFocus = 10.0f;
        float aperture = 0.0f;
        float aspect_ratio = 3.0f / 2.0f;
        float vfov = 40.0f;

        *(d_cam) = new camera(lookFrom, lookAt, vec3(0.0f, 1.0f, 0.0f), vfov, aspect_ratio, aperture, distToFocus, 0.0f, 1.0f);
    }
}

__global__ void Render::freeWorld(hittable **d_list, hittable **d_world, camera **d_cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < NODE_COUNT; i++) {
            delete *(d_list + i);
        }
        delete *(d_world);
        delete *(d_cam);
    }
}

Render::Render(GLuint nx, GLuint ny, hipGraphicsResource_t cuda_pbo_resource) {
    _nx = nx;
    _ny = ny;
    _cuda_pbo_resource = cuda_pbo_resource;

    GLint num_pixels = _nx * _ny;

    hipError_t cudaStatus;

    // create random state for each pixel
    hiprandState *d_randState;
    cudaStatus = hipMalloc((void**)&d_randState, num_pixels * sizeof(hiprandState));
    checkReturn(cudaStatus);

    // create random state for world construction
    hiprandState *d_worldRandState;
    cudaStatus = hipMalloc((void**)&d_worldRandState, sizeof(hiprandState));
    checkReturn(cudaStatus);

    // create world of hittable objects
    hittable **d_list;
    cudaStatus = hipMalloc((void**)&d_list, NODE_COUNT * sizeof(hittable*));
    checkReturn(cudaStatus);

    hittable **d_world;
    cudaStatus = hipMalloc((void**)&d_world, sizeof(hittable*));
    checkReturn(cudaStatus);

    // create camera
    camera **d_cam;
    cudaStatus = hipMalloc((void**)&d_cam, sizeof(camera*));
    checkReturn(cudaStatus);

    dim3 blockCount(_nx + TX - 1 / TX, _ny + TY - 1 / TY);
    dim3 blockSize(TX, TY);

    renderInit<<<blockCount, blockSize>>>(_nx, _ny, d_randState, d_worldRandState);
    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    allocateWorld<<<1, 1>>>(d_list, d_world, d_cam, d_worldRandState);
    checkReturn(hipGetLastError());

    _d_cam = d_cam;
    _d_world = d_world;
    _d_randStatePixels = d_randState;

    hipFree(d_worldRandState);
    hipFree(d_list);
}

__host__ void Render::render() {
    dim3 blockCount(_nx + TX - 1 / TX, _ny + TY - 1 / TY);
    dim3 blockSize(TX, TY);
    GLuint ns = 5;

    // Get the pointer to the frame buffer
    vec3 *fb;
    size_t size;
    hipError_t cudaStatus;
    
    cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&fb, &size, _cuda_pbo_resource);
    checkReturn(cudaStatus);
    
    raytrace<<<blockCount, blockSize>>>(fb, _nx, _ny, ns, _d_cam, _d_world, _d_randStatePixels);
    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    // Unmap the resource
    cudaStatus = hipGraphicsUnmapResources(1, &_cuda_pbo_resource);
    checkReturn(cudaStatus);
}

__host__ void Render::free() {
    hipError_t cudaStatus;

    // free world of hittable objects
    freeWorld<<<1, 1>>(_d_list, _d_world, _d_cam);
    checkReturn(hipGetLastError());

    cudaStatus = hipFree(_d_list);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(_d_world);
    checkReturn(cudaStatus);
}
