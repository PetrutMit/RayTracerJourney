#include "hip/hip_runtime.h"
/* Main file to Ray Trace a scene with multiple spheres
 * Still testing to find the best way to allocate CUDA memory
 * for this
*/

#include "header.cuh"

__device__ color rayColor(const ray& r, hittable **world, hiprandState *localRandState) {
   ray curRay = r;

   float curAttenuation = 1.0f;

   for (int i = 0; i < 50; i ++) {
        hit_record rec;
        if ((*world)->hit(curRay, 0.001f, FLT_MAX, rec)) {
            vec3 target = rec.p + rec.normal + randomInHemiSphere(rec.normal, localRandState);
            curAttenuation *= 0.5f;
            curRay = ray(rec.p, target - rec.p);
        } else {
            vec3 unitDirection = unit_vector(curRay.direction());
            float t = 0.5f * (unitDirection.y() + 1.0f);
            color c1(1.0f, 1.0f, 1.0f);
            color c2(0.5f, 0.7f, 1.0f);
            return curAttenuation * ((1.0f - t) * c1 + t * c2);
        }
   }
   // exceeded recursion
   return color(0.0f, 0.0f, 0.0f);
}

__global__ void renderInit(int maxX, int maxY, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    int pixelIndex = j * maxX + i;

    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixelIndex, 0, &randState[pixelIndex]);
}

__global__ void render(vec3 *fb, int maxX, int maxY, int ns, camera **cam, hittable **world, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    int pixelIndex = j * maxX + i;
    hiprandState localRandState = randState[pixelIndex];
    color pixelColor(0.0f, 0.0f, 0.0f);
    for (int s = 0; s < ns; s ++) {
        float u = float(i + hiprand_uniform(&localRandState)) / float(maxX);
        float v = float(j + hiprand_uniform(&localRandState)) / float(maxY);

        ray r = (*cam)->get_ray(u, v);
        pixelColor += rayColor(r, world, &localRandState);
    }

    getColor(pixelColor, ns);

    fb[pixelIndex] = pixelColor;
}

__global__ void allocateWorld(hittable **d_list, hittable **d_world, camera **d_cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_world) = new hittable_list(d_list, 2);
        *(d_cam) = new camera();

        // ground sphere + big sphere
        *(d_list)   = new sphere(vec3(0,0,-1), 0.5);
        *(d_list+1) = new sphere(vec3(0,-100.5,-1), 100);
    }
}

__global__ void freeWorld(hittable **d_list, hittable **d_world, camera **d_cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        delete *(d_list);
        delete *(d_list+1);
        delete *(d_world);
        delete *(d_cam);
    }
}

int main(void) {
    int nx = 1200;
    int ny = 600;
    int ns = 100;

    int num_pixels = nx * ny;

    color *fb_gpu;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&fb_gpu, num_pixels * sizeof(color));
    checkReturn(cudaStatus);

    // create world of hittable objects
    hittable **d_list;
    cudaStatus = hipMalloc((void**)&d_list, 2 * sizeof(hittable*));
    checkReturn(cudaStatus);

    hittable **d_world;
    cudaStatus = hipMalloc((void**)&d_world, sizeof(hittable*));
    checkReturn(cudaStatus);

    // create camera
    camera **d_cam;
    cudaStatus = hipMalloc((void**)&d_cam, sizeof(camera*));

    allocateWorld<<<1, 1>>>(d_list, d_world, d_cam);
    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    // create random state
    hiprandState *d_randState;
    cudaStatus = hipMalloc((void**)&d_randState, num_pixels * sizeof(hiprandState));
    checkReturn(cudaStatus);


    dim3 blockCount(nx / TX + 1, ny / TY + 1);
    dim3 blockSize(TX, TY);

    renderInit<<<blockCount, blockSize>>>(nx, ny, d_randState);

    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    render<<<blockCount, blockSize>>>(fb_gpu, nx, ny, ns, d_cam, d_world, d_randState);

    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    color *fb_cpu = (color*)malloc(num_pixels * sizeof(color));
    cudaStatus = hipMemcpy(fb_cpu, fb_gpu, num_pixels * sizeof(color), hipMemcpyDeviceToHost);
    checkReturn(cudaStatus);

    // Output FB as Image
    std::ofstream ppmFile("diffuse_spheres.ppm");

    ppmFile << "P3\n" << nx << " " << ny << "\n255\n";

    for (int j = ny - 1; j >= 0; j--) {
        std::cerr << "\rScanlines remaining: " << j << " " << std::flush;
        for (int i = 0; i < nx; i++) {
            size_t pixelIndex = j * nx + i;
            int ir = static_cast<int>(fb_cpu[pixelIndex].e[0]);
            int ig = static_cast<int>(fb_cpu[pixelIndex].e[1]);
            int ib = static_cast<int>(fb_cpu[pixelIndex].e[2]);
            ppmFile << ir << " " << ig << " " << ib << "\n";
    }
}

    // free world of hittable objects
    freeWorld<<<1, 1>>>(d_list, d_world, d_cam);
    checkReturn(hipGetLastError());

    cudaStatus = hipFree(d_list);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(d_world);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(fb_gpu);
    checkReturn(cudaStatus);

    free(fb_cpu);
    std::cerr << "\nDone.\n";
}