#include "hip/hip_runtime.h"
// /* Main file to Ray Trace a scene with multiple spheres
//  * Still testing to find the best way to allocate CUDA memory
//  * for this
// */

#include "header.cuh"

#define SPHERE_COUNT 488

__device__ color rayColor(const ray& r, hittable **world, hiprandState *localRandState) {
   ray curRay = r;

    color curAttenuation(1.0f, 1.0f, 1.0f);

   for (int i = 0; i < 50; i ++) {
        hit_record rec;
        if ((*world)->hit(curRay, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            color attenuation;
            if (rec.mat_ptr->scatter(curRay, rec, attenuation, scattered, localRandState)) {
                curAttenuation = curAttenuation * attenuation;
                curRay = scattered;
            } else {
                return color(0.0f, 0.0f, 0.0f);
            }
       } else {
            vec3 unitDirection = unit_vector(curRay.direction());
            float t = 0.5f * (unitDirection.y() + 1.0f);
            color c1(1.0f, 1.0f, 1.0f);
            color c2(0.5f, 0.7f, 1.0f);
            return curAttenuation * ((1.0f - t) * c1 + t * c2);
        }
   }
   // exceeded recursion
   return color(0.0f, 0.0f, 0.0f);
}

__global__ void renderInit(int maxX, int maxY, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    int pixelIndex = j * maxX + i;

    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixelIndex, 0, &randState[pixelIndex]);
}

__global__ void randInit(hiprandState *randState) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, randState);
    }
}

__global__ void render(vec3 *fb, int maxX, int maxY, int ns, camera **cam, hittable **world, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= maxX) || (j >= maxY)) return;

    int pixelIndex = j * maxX + i;
    hiprandState localRandState = randState[pixelIndex];
    color pixelColor(0.0f, 0.0f, 0.0f);
    for (int s = 0; s < ns; s ++) {
        float u = float(i + hiprand_uniform(&localRandState)) / float(maxX);
        float v = float(j + hiprand_uniform(&localRandState)) / float(maxY);

        ray r = (*cam)->get_ray(u, v, &localRandState);
        pixelColor += rayColor(r, world, &localRandState);
    }

    getColor(pixelColor, ns);

    fb[pixelIndex] = pixelColor;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void allocateWorld(hittable **d_list, hittable **d_world, camera **d_cam, hiprandState *d_randState) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_world) = new hittable_list(d_list,SPHERE_COUNT);

        hiprandState localRandState = *d_randState;

        // Spheres
        material *groundMat = new lambertian(vec3(0.5f, 0.5f, 0.5f));
        *(d_list) = new sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f, groundMat);
        int cnt = 1;
        for (int a = -11; a < 11; a ++) {
            for (int b = -11; b < 11; b ++) {
                float chooseMat = hiprand_uniform(&localRandState);
                vec3 center(a + hiprand_uniform(&localRandState), 0.2f, b + hiprand_uniform(&localRandState));
                material *sphereMat;
                    if (chooseMat < 0.8f) {
                        // diffuse
                        vec3 albedo = vec3(hiprand_uniform(&localRandState) * hiprand_uniform(&localRandState),
                                           hiprand_uniform(&localRandState) * hiprand_uniform(&localRandState),
                                           hiprand_uniform(&localRandState) * hiprand_uniform(&localRandState));
                        sphereMat = new lambertian(albedo);
                        *(d_list + cnt++) = new sphere(center, 0.2f, sphereMat);
                    } else if (chooseMat < 0.95f) {
                        // metal
                        vec3 albedo = vec3(0.5f * (1.0f + hiprand_uniform(&localRandState)),
                                           0.5f * (1.0f + hiprand_uniform(&localRandState)),
                                           0.5f * (1.0f + hiprand_uniform(&localRandState)));
                        float fuzz = 0.5f * hiprand_uniform(&localRandState);
                        sphereMat = new metal(albedo, fuzz);
                        *(d_list + cnt++) = new sphere(center, 0.2f, sphereMat);
                    } else {
                        // glass
                        sphereMat = new dielectric(1.5f);
                        *(d_list + cnt++) = new sphere(center, 0.2f, sphereMat);
                    }
            }
        }
        material *mat1 = new dielectric(1.5f);
        *(d_list+ cnt++) = new sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, mat1);

        material *mat2 = new lambertian(vec3(0.4f, 0.2f, 0.1f));
        *(d_list+ cnt++) = new sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, mat2);

        material *mat3 = new metal(vec3(0.7f, 0.6f, 0.5f), 0.0f);
        *(d_list+ cnt++) = new sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, mat3);


        // Camera

        vec3 lookFrom(13.0f, 2.0f, 3.0f);
        vec3 lookAt(0.0f, 0.0f, 0.0f);
        vec3 vUp(0.0f, 1.0f, 0.0f);

        float distToFocus = 10.0f;
        float aperture = 0.1f;
        float aspect_ratio = 3.0f / 2.0f;

        *d_cam = new camera(lookFrom, lookAt, vUp, 20.0f, aspect_ratio, aperture, distToFocus);
    }
}

__global__ void freeWorld(hittable **d_list, hittable **d_world, camera **d_cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < SPHERE_COUNT; i ++) {
            delete ((sphere *)d_list[i])->mat_ptr;
            delete *(d_list + i);
        }
        delete *(d_world);
        delete *(d_cam);
    }
}

int main(void) {
    int nx = 1200;
    int ny = 800;
    int ns = 10;

    int num_pixels = nx * ny;

    color *fb_gpu;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&fb_gpu, num_pixels * sizeof(color));
    checkReturn(cudaStatus);

    // create random state for each pixel
    hiprandState *d_randState;
    cudaStatus = hipMalloc((void**)&d_randState, num_pixels * sizeof(hiprandState));
    checkReturn(cudaStatus);

    // create random state for world construction
    hiprandState *d_worldRandState;
    cudaStatus = hipMalloc((void**)&d_worldRandState, sizeof(hiprandState));
    checkReturn(cudaStatus);

    // create world of hittable objects
    hittable **d_list;
    cudaStatus = hipMalloc((void**)&d_list, 488 * sizeof(hittable*));
    checkReturn(cudaStatus);

    hittable **d_world;
    cudaStatus = hipMalloc((void**)&d_world, sizeof(hittable*));
    checkReturn(cudaStatus);

    // create camera
    camera **d_cam;
    cudaStatus = hipMalloc((void**)&d_cam, sizeof(camera*));
    checkReturn(cudaStatus);

    randInit<<<1, 1>>>(d_worldRandState);
    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    allocateWorld<<<1, 1>>>(d_list, d_world, d_cam, d_worldRandState);
    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    dim3 blockCount(nx / TX + 1, ny / TY + 1);
    dim3 blockSize(TX, TY);

    renderInit<<<blockCount, blockSize>>>(nx, ny, d_randState);

    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    render<<<blockCount, blockSize>>>(fb_gpu, nx, ny, ns, d_cam, d_world, d_randState);

    checkReturn(hipGetLastError());
    checkReturn(hipDeviceSynchronize());

    color *fb_cpu = (color*)malloc(num_pixels * sizeof(color));
    cudaStatus = hipMemcpy(fb_cpu, fb_gpu, num_pixels * sizeof(color), hipMemcpyDeviceToHost);
    checkReturn(cudaStatus);

    // Output FB as Image
    std::ofstream ppmFile("final_scene.ppm");

    ppmFile << "P3\n" << nx << " " << ny << "\n255\n";

    for (int j = ny - 1; j >= 0; j--) {
        std::cerr << "\rScanlines remaining: " << j << " " << std::flush;
        for (int i = 0; i < nx; i++) {
            size_t pixelIndex = j * nx + i;
            int ir = static_cast<int>(fb_cpu[pixelIndex].e[0]);
            int ig = static_cast<int>(fb_cpu[pixelIndex].e[1]);
            int ib = static_cast<int>(fb_cpu[pixelIndex].e[2]);
            ppmFile << ir << " " << ig << " " << ib << "\n";
    }
}

    // free world of hittable objects
    freeWorld<<<1, 1>>>(d_list, d_world, d_cam);
    checkReturn(hipGetLastError());

    cudaStatus = hipFree(d_list);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(d_world);
    checkReturn(cudaStatus);

    cudaStatus = hipFree(fb_gpu);
    checkReturn(cudaStatus);

    free(fb_cpu);
    std::cerr << "\nDone.\n";
}
